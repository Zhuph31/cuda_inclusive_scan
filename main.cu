#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// CAUTION: DO NOT MODIFY OR SUBMIT THIS FILE
////////////////////////////////////////////////////////////////////////////////

#include <algorithm>
#include <cuda_device_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>
#include <unistd.h>
#include <vector>

#include "implementation.h"
#include "reference_implementation.h"
#include "sampler.h"
#include "util.h"
#include "util_gpu_err_check.cuh"

void printHelp(char *argv[]) {
  std::cout << "Usage: " << argv[0] << " [-g] [-h]\n";
  std::cout << "Options:\n";
  std::cout << "  -g    Set grade mode\n";
  std::cout << "  -h    Display this help message\n";
}

int main(int argc, char *argv[]) {

  //   // get gpu info
  //   hipSharedMemConfig pConfig =
  //       hipSharedMemConfig::hipSharedMemBankSizeDefault;
  //   hipDeviceGetSharedMemConfig(&pConfig);
  //   printf("showing gpu info:\n");
  //   printf("num bank:%d\n", pConfig);
  //   hipDeviceSetSharedMemConfig(
  //       hipSharedMemConfig::hipSharedMemBankSizeEightByte);

  bool grade_mode = false;
  bool exclusive = false, print = false;

  int opt;
  while ((opt = getopt(argc, argv, "ghep")) != -1) {
    switch (opt) {
    case 'g':
      grade_mode = true;
      break;
    case 'h':
      printHelp(argv);
      return 0;
    case 'e':
      exclusive = true;
      break;
    case 'p':
      print = true;
      break;
    default:
      std::cerr << "Unknown option: " << opt << "\n";
      printHelp(argv);
      return 1;
    }
  }

  printSubmissionInfo();
  printf("*********************************************************************"
         "**********************************\n");

  /* generate input */
  constexpr size_t input_size = 100000007u;
  // constexpr size_t input_size = 10;
  std::vector<int32_t> input(input_size);
  std::vector<int32_t> reference_output(input_size);
  std::vector<int32_t> student_output(input_size);
  generateInput(input.data(), input_size);

  int32_t *d_input, *d_output;
  gpu_err_check(hipMalloc((void **)&d_input, input_size * sizeof(int32_t)));
  gpu_err_check(hipMalloc((void **)&d_output, input_size * sizeof(int32_t)));
  gpu_err_check(hipMemcpy(d_input, input.data(), input_size * sizeof(int32_t),
                           hipMemcpyHostToDevice));

  hipStream_t stream;
  gpu_err_check(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  if (grade_mode) {
    Sampler sampler;
    std::cout << "Performance Results:" << std::endl;

    uint32_t reference_time =
        sampler.sample(referenceImplementation, input.data(),
                       reference_output.data(), input_size, exclusive);
    std::cout << "\tTime consumed by the sequential implementation: "
              << reference_time << "us" << std::endl;

    uint32_t student_time =
        sampler.sample(implementation, d_input, d_output, input_size);
    std::cout << "\tTime consumed by your implementation: " << student_time
              << "us" << std::endl;

    std::cout << "\tOptimization Speedup Ratio (nearest integer): "
              << (int)((double)reference_time / std::max(student_time, 1u) +
                       0.5)
              << std::endl;
    printf("*******************************************************************"
           "************************************"
           "\n");
  }

  /* verify results */
  std::fill(reference_output.begin(), reference_output.end(), 0);
  std::fill(student_output.begin(), student_output.end(), 0);

  referenceImplementation(input.data(), reference_output.data(), input_size,
                          exclusive);
  implementation(d_input, d_output, input_size);
  gpu_err_check(hipMemcpyAsync(student_output.data(), d_output,
                                input_size * sizeof(int32_t),
                                hipMemcpyDeviceToHost, stream));

  gpu_err_check(hipStreamSynchronize(stream));
  gpu_err_check(hipFree(d_input));
  gpu_err_check(hipFree(d_output));
  gpu_err_check(hipStreamDestroy(stream));

  if (std::equal(reference_output.begin(), reference_output.end(),
                 student_output.begin())) {
    std::cout << "Your implementation is correct." << std::endl;
  } else {
    std::cerr << "Your implementation is incorrect." << std::endl;
    if (print) {
      for (int i = 0; i < input_size; ++i) {
        printf("%d,", input[i]);
      }
      printf("\n");
      for (int i = 0; i < input_size; ++i) {
        printf("%d,", reference_output[i]);
      }
      printf("\n");
      for (int i = 0; i < input_size; ++i) {
        printf("%d,", student_output[i]);
      }
      printf("\n");
    }
    exit(-1);
  }

  return 0;
}